#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#ifdef _WIN32
    #include <Windows.h>
#else
    #include <sys/time.h>
#endif


#define CHECK(func)\
{\
  const hipError_t error = func;\
  if(error != hipSuccess)\
  {\
      printf("ERROR: %s: %d, ", __FILE__, __LINE__);\
      printf("code: %d, reason: %s\n", error, hipGetErrorString(error));\
      exit(EXIT_FAILURE);\
  }\
}

void init_device(int device) {
    int dev = device;
    int deviceCount = 0;
    hipDeviceProp_t deviceProp;

    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    CHECK(hipGetDeviceCount(&deviceCount));
    CHECK(hipSetDevice(dev));
    printf("Set cuda device %d, device name: %s, device count: %d\n", dev, deviceProp.name, deviceCount);
}

void init_data(float *data, int size)
{
    // 随机产生-5到5之间的随机正负数
    time_t t;
    srand((unsigned int)time(&t));

    for (int i = 0; i < size; i++) {
        data[i] = rand() % 11 - 5;
    }
}

// sigmoid function kernel
__global__ void relu(float* input, float* output, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) output[idx] = fmaxf(0.0f, input[idx]);
}

// host code to invoke the kernel
void relu_wrapper(float *h_input, float *h_output, int N)
{
    float *d_input, *d_output;
    size_t memSize = N * sizeof(float);

    CHECK(hipMalloc((void **)&d_input, memSize));
    CHECK(hipMalloc((void **)&d_output, memSize));

    // 将host的input copy到device的input
    CHECK(hipMemcpy(d_input, h_input, memSize, hipMemcpyHostToDevice));

    int blockSize = 256;
    dim3 block(blockSize, 1);
    dim3 grid((N - 1) / block.x + 1, 1);
    relu<<<grid, block>>>(d_input, d_output, N);

    // 将device的output copy到host端
    CHECK(hipMemcpy(h_output, d_output, memSize, hipMemcpyDeviceToHost));

    // free
    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char **argv) 
{
    // device
    init_device(0);

    // data
    int N = argc > 1 ? atoi(argv[1]) : 10;
    size_t memSize = N * sizeof(float);

    float *h_input = (float *)malloc(memSize);
    float *h_output = (float *)malloc(memSize);
    memset(h_input, 0, memSize);
    memset(h_output, 0, memSize);
    init_data(h_input, N);

    // kernel function 
    relu_wrapper(h_input, h_output, N);

    for (size_t i = 0; i < N; i++)
    {
        printf("src val: %f, relu result: %f\n", h_input[i], h_output[i]);
    }

    // free
    free(h_input);
    free(h_output);
}